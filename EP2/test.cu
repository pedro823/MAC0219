#include "hip/hip_runtime.h"
#include "io.hpp"
#include "min.hpp"
#include "error_handler.hpp"
#include <iostream>
#include <ctime>
#include <unistd.h>
#include <sys/time.h>
using namespace std;

int main(int argc, char ** argv) {
    if (argc < 2) {
        fprintf(stderr, "Not enough arguments");
        return 1;
    }

    printf("Reading Matrices\n");

    char * matrixFile = argv[1];
    Matrices a = readMatricesFromFile(matrixFile);
     
    allocateMatricesToCuda(a);

    printf("Matrixes in memory\n");

    printf("there are %d matrices\n", a.length);
    
    struct timeval begin, end;

    bool same_ans = true;
    int elapsed = 0;

    gettimeofday(&begin, NULL);
    int * cudaAns = cudaReduceMatrix(a);
    hipDeviceSynchronize();
    gettimeofday(&end, NULL);
        
    elapsed = (end.tv_sec - begin.tv_sec) * 1000000 + (end.tv_usec - begin.tv_usec);
        
    printf("Cuda ans -- %d microseconds\n", elapsed);
    for (int i = 0; i < 9; i++) {
        printf("%d ", cudaAns[i]);
    }

    gettimeofday(&begin, NULL);
    int * seqAns  = sequentialReductionMatrix(a);
    hipDeviceSynchronize();
    gettimeofday(&end, NULL);
        
    elapsed = (end.tv_sec - begin.tv_sec) * 1000000 + (end.tv_usec - begin.tv_usec);
    
    printf("\n\nSequential ans -- %d microseconds\n", elapsed);
    for (int i = 0; i < 9; i++) {
        printf("%d ", seqAns[i]);
        if (seqAns[i] != cudaAns[i]) same_ans = false;
    }
    printf("\n\n");
    
    printf("%s\n", (same_ans ? "Cuda is CORRECT" : "Cuda is WRONG"));

    return 0;
}
