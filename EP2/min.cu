#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <iostream>
#include <algorithm>
using namespace std;
#include "min.hpp"

int magicMin(int a, int b) {
	return (a < b) ? a : b;
}

int sequentialReductionArray(int * v, int n) {
    int res = v[0];
    for (int i = 1; i < n; i++) {
        res = magicMin(res, v[i]);
    }
    return res;
}

__global__
void cudaReduceArray(int * in, int * out, int n) {
    extern __shared__ int sdata[]; // Array with size BLOCK_NUM

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x * 2 + threadIdx.x;

    int cur, nxt;
    cur = (i < n) ? in[i] : INF;
    nxt = (i + blockDim.x < n) ? in[i + blockDim.x] : INF;
    
    sdata[tid] = min(cur, nxt);

    __syncthreads();    
    
    for (int s = blockDim.x / 2; s > 0; s /= 2) {
        if (tid < s) {
            sdata[tid] = min(sdata[tid], sdata[tid+s]);
        }
        __syncthreads();
    }

    if (tid == 0) out[blockIdx.x] = sdata[0];
}

int * sequentialReductionMatrix(Matrices m) {
    int * res = (int *)malloc(9 * sizeof(int));
    for (int i = 0; i < 9; i++) {
        res[i] = sequentialReductionArray(m.v[i], m.length);
    }
    return res;
}

int* cudaReduceMatrix(Matrices m) {
    int * res = (int *)malloc(9 * sizeof(int));
    const int THREAD_NUM = 32;
    const int BLOCK_NUM =  (m.length + 2 * THREAD_NUM - 1)/ (2 * THREAD_NUM);

    int * cuda_block_results;
    int * host_block_results;
    
    hipMalloc(&cuda_block_results, BLOCK_NUM * sizeof(int));
    host_block_results = (int *)malloc(BLOCK_NUM * sizeof(int));
        
    
    for (int i = 0; i < 9; i++) {

        cudaReduceArray<<<BLOCK_NUM, THREAD_NUM, THREAD_NUM * sizeof(int)>>>(m.dv[i], cuda_block_results, m.length);
        hipDeviceSynchronize();

        hipMemcpy(host_block_results, cuda_block_results, BLOCK_NUM * sizeof(int), hipMemcpyDeviceToHost);

        res[i] = host_block_results[0];
        for (int k = 1; k < BLOCK_NUM; k++) {
            res[i] = min(res[i], host_block_results[k]);            
        }
    }

    hipFree(cuda_block_results);
    free(host_block_results);

    return res;
}
