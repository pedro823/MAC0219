#include <sstream>
#include <fstream>
#include <string>
#include <stdexcept>
#include "io.hpp"
#include "error_handler.hpp"

#define debug asm("int $3")

using namespace std;

Matrices readMatricesFromFile(const char *fileName) {
    string line;
    ifstream infile(fileName);
    // The state of the parser.
    // 0: waiting for number of matrices
    // 1: waiting for matrices numbers
    // 2: waiting for asterisks
    // 3: waiting for EOF
    int state = 0;
    int matrixLine = 0;
    long long fileLine = 0;
    // 2d matrices flattened in a vector
    int ** v = (int **)malloc(sizeof(int *) * 9);
    long long index = 0;
    long long numberOfMatrices, currentMatrix = -1;
    bool doContinue = true;
    stringstream err;
    while (getline(infile, line) && doContinue) {
        fileLine++;
        istringstream it(line);
        switch (state) {
            case 0: {
                long long n;
                if (!(it >> n)) {
                    err << "readMatricesFromFile:" << fileLine
                        << ": Expected number";
                    throw runtime_error(err.str());
                }
                else {
                    numberOfMatrices = n;
                    // 9 numbers per matrix
                    for (int i = 0; i < 9; i++)
                        v[i] = (int *)malloc(sizeof(int) * n);;
                    state = 2;
                }
                break;
            }
            case 1: {
                int a, b, c;
                if (!(it >> a >> b >> c)) {
                    err << "readMatricesFromFile:" << fileLine
                    << ": Expected number";
                    throw runtime_error(err.str());
                }
                else {
                    v[(matrixLine) * 3][currentMatrix] = a;
                    v[(matrixLine) * 3 + 1][currentMatrix] = b;
                    v[(matrixLine) * 3 + 2][currentMatrix] = c;

                    // v[(matrixLine * 3) * numberOfMatrices + currentMatrix] = a;
                    // v[(matrixLine * 3 + 1) * numberOfMatrices + currentMatrix] = b;
                    // v[(matrixLine * 3 + 2) * numberOfMatrices + currentMatrix] = c;

                    index += 3;
                    if (matrixLine == 2) {
                        // go to asterisks
                        matrixLine = 0;
                        state = 2;
                    }
                    else {
                        matrixLine++;
                    }
                }
                break;
            }
            case 2: {
                // asterisks
                string ask;
                if (!(it >> ask) || ask.compare("***") != 0) {
                    err << "readMatricesFromFile:" << fileLine
                        << ": Expected '***', got " << ask;
                    throw runtime_error(err.str());
                }
                if (currentMatrix >= numberOfMatrices - 1) {
                    state = 3;
                }
                else {
                    currentMatrix++;
                    state = 1;
                }
                break;
            }
            case 3: {
                // Throws the rest away.
                doContinue = false;
                break;
            }
            default: {
                err << "readMatricesFromFile:" << fileLine
                    << ": invalid parser state: " << state;
                throw runtime_error(err.str());
            }
        }
    }
    if (state != 3) {
        err << "readMatricesFromFile:" << fileLine
            << ": could not read matrices from " << fileName;
        throw runtime_error(err.str());
    }
    
    Matrices result;
    result.v = (int **)malloc(sizeof(int *) * 9);
    result.dv = (int **)malloc(sizeof(int *) * 9);
    for (int i = 0; i < 9; i++) {
        result.v[i] = v[i];
        result.dv[i] = NULL;
    }
    result.length = numberOfMatrices;
    return result;
}

void allocateMatricesToCuda(Matrices& m) {
    for (int i = 0; i < 9; i++) {
        hipMallocManaged(&(m.dv[i]), m.length * sizeof(int));
        errorCheck();
        hipMemcpy(m.dv[i], m.v[i], m.length * sizeof(int), hipMemcpyHostToDevice);
        errorCheck();
    }
}
