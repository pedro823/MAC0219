#include "error_handler.hpp"
#include <sstream>
#include <stdexcept>

using namespace std;

void cudaErrorHandler(const char *file, int line) {
    hipDeviceSynchronize();
    stringstream err;
    hipError_t lastError = hipGetLastError();
    if (lastError != hipSuccess) {
        err << file << ":" << line <<  ": CudaError: "
            << hipGetErrorString(lastError) << endl;
        throw runtime_error(err.str());
    }
}
